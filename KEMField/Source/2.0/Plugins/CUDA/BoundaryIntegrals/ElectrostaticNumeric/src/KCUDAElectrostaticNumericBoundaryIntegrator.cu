#include <iostream>
#include <fstream>
#include <sstream>

#include "KCUDAElectrostaticNumericBoundaryIntegrator.hh"
#include "KSurfaceTypes.hh"
#include "KCUDASurfaceContainer.hh"


namespace KEMField
{

KCUDAElectrostaticNumericBoundaryIntegrator::KCUDAElectrostaticNumericBoundaryIntegrator(KCUDASurfaceContainer& c) :
		KCUDABoundaryIntegrator<KElectrostaticBasis>(c),
		fDevicePhi(NULL),
		fDeviceEField(NULL),
		fDeviceEFieldAndPhi(NULL)
{
	ConstructCUDAKernels();
	AssignDeviceMemory();
}

KCUDAElectrostaticNumericBoundaryIntegrator::~KCUDAElectrostaticNumericBoundaryIntegrator()
{
	if( fDevicePhi ) hipFree(fDevicePhi);
	if( fDeviceEField ) hipFree(fDeviceEField);
	if( fDeviceEFieldAndPhi ) hipFree(fDeviceEFieldAndPhi);
}

void KCUDAElectrostaticNumericBoundaryIntegrator::BoundaryVisitor::Visit(KDirichletBoundary& boundary)
{
	fIsDirichlet = true;
	fPrefactor = 1.;
	fBoundaryValue = static_cast<DirichletBoundary&>(boundary).GetBoundaryValue();
}

void KCUDAElectrostaticNumericBoundaryIntegrator::BoundaryVisitor::Visit(KNeumannBoundary& boundary)
{
	fIsDirichlet = false;
	fPrefactor = ((1. + static_cast<NeumannBoundary&>(boundary).GetNormalBoundaryFlux())/(1. - static_cast<NeumannBoundary&>(boundary).GetNormalBoundaryFlux()));
	fBoundaryValue = 0.;
}

void KCUDAElectrostaticNumericBoundaryIntegrator::BasisVisitor::Visit(KElectrostaticBasis& basis)
{
	fBasisValue = &(basis.GetSolution());
}

KElectrostaticBasis::ValueType KCUDAElectrostaticNumericBoundaryIntegrator::BoundaryIntegral(KSurfacePrimitive* source,KSurfacePrimitive* target,unsigned int)
{
	fTarget = target;
	target->Accept(fBoundaryVisitor);
	source->Accept(*this);
	return fValue;
}

KElectrostaticBasis::ValueType KCUDAElectrostaticNumericBoundaryIntegrator::BoundaryValue(KSurfacePrimitive* surface,unsigned int)
{
	surface->Accept(fBoundaryVisitor);
	return fBoundaryVisitor.GetBoundaryValue();
}

KElectrostaticBasis::ValueType& KCUDAElectrostaticNumericBoundaryIntegrator::BasisValue(KSurfacePrimitive* surface,unsigned int)
{
	surface->Accept(fBasisVisitor);
	return fBasisVisitor.GetBasisValue();
}

void KCUDAElectrostaticNumericBoundaryIntegrator::ConstructCUDAKernels() const
{
	KCUDABoundaryIntegrator<KElectrostaticBasis>::ConstructCUDAKernels();

	// Create memory buffers

	hipMalloc( (void**) &fDeviceP, 3*sizeof(CU_TYPE) );
	hipMalloc( (void**) &fDeviceShapeInfo, sizeof(short));
	// Hard-coded arbitrary maximum shape limit
	hipMalloc( (void**) &fDeviceShapeData, 20*sizeof(CU_TYPE) );

	hipMalloc( (void**) &fDevicePhi, sizeof(CU_TYPE) );
	hipMalloc( (void**) &fDeviceEField, sizeof(CU_TYPE4) );
	hipMalloc( (void**) &fDeviceEFieldAndPhi, sizeof(CU_TYPE4) );

	// copy weights and nodes to constant memory, get values from CPU integrator class
	// constant variables have been defined in corresponding device functions

    // n.b. CUDA memcopy needed also in executables in order
    // to guarantee that data will be copied onto GPU constant memory

    // 7-point triangle cubature

    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub7alpha), gTriCub7alpha, sizeof(CU_TYPE)*3 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub7beta), gTriCub7beta, sizeof(CU_TYPE)*3 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub7gamma), gTriCub7gamma, sizeof(CU_TYPE)*3 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub7w), gTriCub7w, sizeof(CU_TYPE)*7 );

    // 12-point triangle cubature

    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub12alpha), gTriCub12alpha, sizeof(CU_TYPE)*4 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub12beta), gTriCub12beta, sizeof(CU_TYPE)*4 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub12gamma), gTriCub12gamma, sizeof(CU_TYPE)*4 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub12w), gTriCub12w, sizeof(CU_TYPE)*12 );

    // 33-point triangle cubature

    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub33alpha), gTriCub33alpha, sizeof(CU_TYPE)*8 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub33beta), gTriCub33beta, sizeof(CU_TYPE)*8 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub33gamma), gTriCub33gamma, sizeof(CU_TYPE)*8 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuTriCub33w), gTriCub33w, sizeof(CU_TYPE)*33 );

    // rectangle cubature weights

    hipMemcpyToSymbol(HIP_SYMBOL( cuRectCub7w), gRectCub7w, sizeof(CU_TYPE)*7 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuRectCub12w), gRectCub12w, sizeof(CU_TYPE)*12 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuRectCub33w), gRectCub33w, sizeof(CU_TYPE)*33 );

    // quadrature weights and nodes for line segments

    hipMemcpyToSymbol(HIP_SYMBOL( cuLineQuadx4), gQuadx4, sizeof(CU_TYPE)*2 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuLineQuadw4), gQuadw4, sizeof(CU_TYPE)*2 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuLineQuadx16), gQuadx16, sizeof(CU_TYPE)*8 );
    hipMemcpyToSymbol(HIP_SYMBOL( cuLineQuadw16), gQuadw16, sizeof(CU_TYPE)*8 );
}

void KCUDAElectrostaticNumericBoundaryIntegrator::AssignDeviceMemory() const
{
	KCUDABoundaryIntegrator<KElectrostaticBasis>::AssignDeviceMemory();
}

}
